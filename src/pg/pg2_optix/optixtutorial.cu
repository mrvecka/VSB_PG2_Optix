#include "hip/hip_runtime.h"
#include "optixtutorial.h"

struct TriangleAttributes
{
	optix::float3 normal;
	optix::float2 texcoord;
	optix::float3 intersectionPoint;
	optix::float3 vectorToLight;
};

rtBuffer<optix::float3, 1> normal_buffer;
rtBuffer<optix::float2, 1> texcoord_buffer;
rtBuffer<optix::uchar4, 2> output_buffer;

rtDeclareVariable(optix::float3, diffuse, , "diffuse");
rtDeclareVariable(optix::float3, specular, , "specular");
rtDeclareVariable(optix::float3, ambient, , "ambient");
rtDeclareVariable(float, shininess, , "shininess");

rtDeclareVariable(int, tex_diffuse_id, , "diffuse texture id");

rtDeclareVariable( rtObject, top_object, , );
rtDeclareVariable( uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( PerRayData_radiance, ray_data, rtPayload, );
rtDeclareVariable(PerRayData_shadow, shadow_ray_data, rtPayload, );
rtDeclareVariable( float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(TriangleAttributes, attribs, attribute attributes, "Triangle attributes");
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(optix::float3, view_from, , );
rtDeclareVariable(optix::Matrix3x3, M_c_w, , "camera to worldspace transformation matrix" );
rtDeclareVariable(float, focal_length, , "focal length in pixels" );

RT_PROGRAM void attribute_program(void)
{
	const optix::float3 lightPossition = optix::make_float3(100, 100, 200);
	const optix::float2 barycentrics = rtGetTriangleBarycentrics();
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];

	const optix::float2 t0 = texcoord_buffer[index * 3 + 0];
	const optix::float2 t1 = texcoord_buffer[index * 3 + 1];
	const optix::float2 t2 = texcoord_buffer[index * 3 + 2];

	attribs.normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));
	attribs.texcoord = t1 * barycentrics.x + t2 * barycentrics.y + t0 * (1.0f - barycentrics.x - barycentrics.y);

	if (optix::dot(ray.direction, attribs.normal) > 0) {
		attribs.normal *= -1;
	}

	attribs.intersectionPoint = optix::make_float3(ray.origin.x + ray.tmax * ray.direction.x,
		ray.origin.y + ray.tmax * ray.direction.y,
		ray.origin.z + ray.tmax * ray.direction.z);

	attribs.vectorToLight = lightPossition - attribs.intersectionPoint;
}

RT_PROGRAM void primary_ray( void )
{

	const optix::float3 d_c = make_float3(launch_index.x -
		launch_dim.x * 0.5f, output_buffer.size().y * 0.5f -
		launch_index.y, -focal_length);
	const optix::float3 d_w = optix::normalize(M_c_w * d_c);
	optix::Ray ray(view_from, d_w, 0, 0.01f);

	PerRayData_radiance prd;
	hiprand_init(launch_index.x + launch_dim.x * launch_index.y, 0, 0, &prd.state);
	rtTrace( top_object, ray, prd );

	// access to buffers within OptiX programs uses a simple array syntax	
	output_buffer[launch_index] = optix::make_uchar4( prd.result.x*255.0f, prd.result.y*255.0f, prd.result.z*255.0f, 255 );
}

RT_PROGRAM void closest_hit_Phong( void )
{
	optix::float3 amb_occ = ambient_occlusion(ray_data.state);
	float ligth = optix::dot(optix::normalize(attribs.vectorToLight), attribs.normal);

	optix::float3 lr = 2 * (ligth)* attribs.normal - optix::normalize(attribs.vectorToLight);
	float shade = shadow_ray(attribs.vectorToLight);
	
	optix::float3 res = ambient + (getDiffuseColor() * ligth) + specular * pow(optix::dot(-ray.direction, lr), shininess);

	ray_data.result = res * amb_occ;

}

RT_PROGRAM void closest_hit_Normal(void)
{
	optix::float3 amb = ambient_occlusion(ray_data.state);
	ray_data.result = attribs.normal * amb * 0.5f;

}

RT_PROGRAM void closest_hit_Lambert(void)
{
	optix::float3 diff = getDiffuseColor();
	float ligth = optix::dot(optix::normalize(attribs.vectorToLight), attribs.normal);

	optix::float3 lr = 2 * (ligth)* attribs.normal - optix::normalize(attribs.vectorToLight);
	optix::float3 res = optix::fmaxf(0, ligth) * diff;
	float shade = shadow_ray(attribs.vectorToLight);
	optix::float3 amb = ambient_occlusion(ray_data.state);
	ray_data.result = res *amb;

}

RT_PROGRAM void any_hit(void)
{
	rtTerminateRay();
}

RT_PROGRAM void shader_hit(void)
{
	shadow_ray_data.visible = 0.0f;
	rtTerminateRay();
}

__device__ optix::float3 ambient_occlusion(hiprandState_t state)
{
	optix::float3 sum = optix::make_float3(0, 0, 0);
	int N = 32;
	for (int i = 0; i < N; i++)
	{
		float randomX = (float)hiprand_uniform(&state);
		float randomY = (float)hiprand_uniform(&state);

		optix::float3 omegai = SampleHemisphere(attribs.normal, randomX,randomY);
		float pdf = 1.0f/ (2* HIP_PI_F);

		float shade = shadow_ray(omegai);

		optix::float3 whiteColor = optix::make_float3(1, 1, 1);
		
		sum += diffuse * shade * (optix::dot(attribs.normal, omegai) / pdf);
	}
	return sum/N;
}

/* may access variables declared with the rtPayload semantic in the same way as closest-hit and any-hit programs */
RT_PROGRAM void miss_program( void )
{
	ray_data.result = optix::make_float3( 0.0f, 0.0f, 0.0f );
}

RT_PROGRAM void exception( void )
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y );
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}

__device__ optix::float3 getDiffuseColor()
{
	optix::float3 color;
	if (tex_diffuse_id != -1) {
		const optix::float4 value = optix::rtTex2D<optix::float4>(tex_diffuse_id, attribs.texcoord.x, 1 - attribs.texcoord.y);
		color = optix::make_float3(value.x, value.y, value.z);
	}
	else {
		color = diffuse;
	}

	return color;
}

__device__ float shadow_ray(optix::float3 dir)
{
	float L = L2Norm(attribs.vectorToLight);

	optix::Ray ray(attribs.intersectionPoint, dir, 1, 0.01f);

	PerRayData_shadow shadow_prd;
	shadow_prd.visible = 1.0f;
	rtTrace(top_object, ray, shadow_prd);

	return shadow_prd.visible;
}

__device__ float L2Norm(optix::float3 q)
{
	return sqrt(q.x * q.x + q.y * q.y + q.z * q.z);
	
}

__device__ optix::float3 SampleHemisphere(optix::float3 normal, float randomX, float randomY)
{
	float x = 2 * cosf(2 * HIP_PI_F * randomX) * sqrtf(randomY * (1 - randomY));
	float y = 2 * sinf(2 * HIP_PI_F * randomX) * sqrtf(randomY * (1 - randomY));
	float z = 1 - 2 * randomY;
	optix::float3 omegaI = optix::make_float3( x, y, z );
	optix::normalize(omegaI);
	if (optix::dot(omegaI, normal) < 0) {
		omegaI *= -1;
	}

	return omegaI;
}

